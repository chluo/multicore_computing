#include <stdio.h>
#include <stdlib.h>
#include <math.h> 
#include <hip/hip_runtime.h>

#define MAX_ARRAY_SIZE 1000000

/* -------------------------------------------------------------------------
  Algorithm description: 
  
  array_in      {1, 5, 3, 2, 6, 7, 9, 5, 3, 6} 
     |
     |  parallel check odd/even: O(1) 
    \_/
  array_is_odd  {1, 1, 1, 0, 0, 1, 1, 1, 1, 0} 
     |
     |  inclusive prefix scan: O(logN) 
    \_/
  array_index   {1, 2, 3, 3, 3, 4, 5, 6, 7, 7}
  
  num_odd := array_index[N - 1];                       -\
  array_o[num_odd];                                      |
  for i = 1 : N - 1 in parallel                          | O(1)
    if array_is_odd[i]                                   | 
        array_o[array_index[i] - 1] = array_in[i];       |
                                                       -/
  array_o       {1, 5, 3, 7, 9, 5, 3}
  
  T(N) = O(logN) 
------------------------------------------------------------------------- */

/* 
* Check GPU device 
*/ 
void check_dev(void) {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        printf("!! Error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }
    int dev = 0;
    hipSetDevice(dev);
}

/*
* Calculate the number of threads per block based on array size 
*/ 
int calc_num_thread(int size) {
    int approx = (int)sqrt((double)size); 
    // find the nearest power of 2 
    int pow2 = 1; 
    while (pow2 < approx) {
        pow2 <<= 1; 
    }
    return pow2; 
}

/* 
* Read data from ./inp.txt 
* Return the pointer to the data array 
* Ouput the number of data items thru passed-in pointer (int * size)
*/ 
int * read_data(int * size) 
{
    FILE * fptr = fopen("./inp_long.txt", "r"); 
    if (!fptr) {
        printf("!! Error in opening data file \n"); 
        exit(1); 
    }
    int cur_array_size = MAX_ARRAY_SIZE; 
    int * buffer = (int *)malloc(cur_array_size * sizeof(int)); 
    
    int i = 0; 
    while (!feof(fptr)) {
        if (fscanf(fptr, "%d, ", &buffer[i]) != 1) {
            break; 
        }
        ++i; 
    }
    
    fclose(fptr); 
    *size = i; 
    return buffer; 
}

/* 
* Outputs the result array into file 
*/ 
void print_file(int * array, int array_size, const char file_name[]) {
    FILE * fptr_b = fopen(file_name, "w"); 
    if (!fptr_b) {
        printf("!! Error in opening output file \n"); 
        exit(1);
    }
    for (int i = 0; i < array_size; ++i) {
        fprintf(fptr_b, "%d", array[i]); 
        if (i < array_size - 1) 
            fprintf(fptr_b, ", "); 
    }
    fclose(fptr_b); 
}

/* 
* GPU kernel: parallel odd/even check
* The output array has 1/odd or 0/even at the corresponding spot
*/ 
__global__ void odd_check(int * array_i, int * array_o, int array_size) {
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    if (myId < array_size) {
        array_o[myId] = array_i[myId] % 2;  
    }
}

/* 
* GPU kernel: inclusive prefix scan, one step 
*/ 
__global__ void prefix_scan_step(int * array_i, int * array_o, int array_size, int dist) {
    // shared memory to store intermediate results 
    extern __shared__ int sdata[]; 
    
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int thId = threadIdx.x; 
    
    // load initial values to shared memory 
    sdata[thId] = array_i[myId]; 
    __syncthreads(); 
    
    // store block results in shared memory 
    if (!(myId < dist) && myId < array_size) {
        sdata[thId] += array_i[myId - dist]; 
    }
    __syncthreads();  
    // copy results to global memory 
    if (myId < array_size) {
        array_o[myId] = sdata[thId]; 
    }
}

/* 
* GPU kernel: inclusive prefix scan, copy result of one step to the input of the next step  
*/ 
__global__ void prefix_scan_copy(int * array_i, int * array_o, int array_size, int dist) {
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    if (myId < array_size) {
        array_i[myId] = array_o[myId]; 
    }
} 

/* 
* Inclusive prefix scan
*/ 
void prefix_scan(int * array_i, int * array_o, int array_size) {
    // dynamically calculate the number of threads and blocks 
    const int maxThreadsPerBlock = calc_num_thread(array_size);
    int threads = maxThreadsPerBlock;
    int blocks = (array_size + maxThreadsPerBlock - 1) / maxThreadsPerBlock;
    
    int dist = 1; 
    while (dist < array_size) {
        prefix_scan_step<<<blocks, threads, threads * sizeof(int)>>>(array_i, array_o, array_size, dist); 
        hipDeviceSynchronize(); 
        prefix_scan_copy<<<blocks, threads, threads * sizeof(int)>>>(array_i, array_o, array_size, dist); 
        hipDeviceSynchronize(); 
        // cudaMemcpy(array_i, array_o, array_size * sizeof(int), cudaMemcpyDeviceToDevice); 
        
        /* 
        if (dist == 1) {
            int * debug = (int *)malloc(array_size * sizeof(int)); 
            cudaMemcpy(debug, array_io, array_size * sizeof(int), cudaMemcpyDeviceToHost); 
            print_file(debug, array_size, "./debug1.txt"); 
            free(debug); 
        }
        if (dist == 2) {
            int * debug = (int *)malloc(array_size * sizeof(int)); 
            cudaMemcpy(debug, array_io, array_size * sizeof(int), cudaMemcpyDeviceToHost); 
            print_file(debug, array_size, "./debug2.txt"); 
            free(debug); 
        }        
        if (dist == 4) {
            int * debug = (int *)malloc(array_size * sizeof(int)); 
            cudaMemcpy(debug, array_io, array_size * sizeof(int), cudaMemcpyDeviceToHost); 
            print_file(debug, array_size, "./debug4.txt"); 
            free(debug); 
        }
        if (dist == 8) {
            int * debug = (int *)malloc(array_size * sizeof(int)); 
            cudaMemcpy(debug, array_io, array_size * sizeof(int), cudaMemcpyDeviceToHost); 
            print_file(debug, array_size, "./debug8.txt"); 
            free(debug); 
        }     
        if (dist == 16) {
            int * debug = (int *)malloc(array_size * sizeof(int)); 
            cudaMemcpy(debug, array_io, array_size * sizeof(int), cudaMemcpyDeviceToHost); 
            print_file(debug, array_size, "./debug16.txt"); 
            free(debug); 
        }   
        if (dist == 32) {
            int * debug = (int *)malloc(array_size * sizeof(int)); 
            cudaMemcpy(debug, array_io, array_size * sizeof(int), cudaMemcpyDeviceToHost); 
            print_file(debug, array_size, "./debug32.txt"); 
            free(debug); 
        }      
        if (dist == 64) {
            int * debug = (int *)malloc(array_size * sizeof(int)); 
            cudaMemcpy(debug, array_io, array_size * sizeof(int), cudaMemcpyDeviceToHost); 
            print_file(debug, array_size, "./debug64.txt"); 
            free(debug); 
        }     
        if (dist == 128) {
            int * debug = (int *)malloc(array_size * sizeof(int)); 
            cudaMemcpy(debug, array_io, array_size * sizeof(int), cudaMemcpyDeviceToHost); 
            print_file(debug, array_size, "./debug128.txt"); 
            free(debug); 
        }  
        if (dist == 262144) {
            int * debug = (int *)malloc(array_size * sizeof(int)); 
            cudaMemcpy(debug, array_io, array_size * sizeof(int), cudaMemcpyDeviceToHost); 
            print_file(debug, array_size, "./debug262144.txt"); 
            free(debug); 
        }        
        if (dist == 524288) {
            int * debug = (int *)malloc(array_size * sizeof(int)); 
            cudaMemcpy(debug, array_io, array_size * sizeof(int), cudaMemcpyDeviceToHost); 
            print_file(debug, array_size, "./debug524288.txt"); 
            free(debug); 
        }
        */ 
        
        dist *= 2; 
    }
}

/* 
* GPU kernel: compact the input array to get the odd numbers 
*/ 
__global__ void get_odd(int * array_i, int * array_o, /* int * array_is_odd, */ int * array_index, int array_size/* , int num_odd */) {
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    if (myId < array_size) {
        if (/* array_is_odd[myId] */ (myId == 0 && myId > 0) || (array_index[myId] > array_index[myId - 1])) {
            array_o[array_index[myId] - 1] = array_i[myId]; 
        }
    }
}

/* 
* Compact algorithm: put the odd numbers in the input array into the output array
* Returns the pointer to the output array
* Ouputs the number of odd numbers thru passed-in pointer (int * num_odd) 
*/ 
int * compact(int * array_i, int * num_odd, int array_size) {
    // dynamically calculate the number of threads and blocks 
    const int maxThreadsPerBlock = calc_num_thread(array_size);
    int threads = maxThreadsPerBlock;
    int blocks = (array_size + maxThreadsPerBlock - 1) / maxThreadsPerBlock;
    
    // copy the input array into GPU shared memory 
    int * array_device; 
    hipMalloc((void **) &array_device, array_size * sizeof(int)); 
    hipMemcpy(array_device, array_i, array_size * sizeof(int), hipMemcpyHostToDevice); 
    
    // allocate GPU memories for array_is_odd and array_index 
    int * array_is_odd, * array_index; 
    hipMalloc((void **) &array_is_odd, array_size * sizeof(int)); 
    hipMalloc((void **) &array_index, array_size * sizeof(int)); 
    
    // compute array_is_odd 
    odd_check<<<blocks, threads>>>(array_device, array_is_odd, array_size); 
    hipDeviceSynchronize();   
    
    // TODO: debug 
    // printf("%s\n", cudaGetErrorString(cudaPeekAtLastError()));
    int * debug = (int *)malloc(array_size * sizeof(int)); 
    hipMemcpy(debug, array_is_odd, array_size * sizeof(int), hipMemcpyDeviceToHost); 
    print_file(debug, array_size, "./debug.txt"); 
    free(debug);     
        
    // populate array_index with initial values  
    hipMemcpy(array_index, array_is_odd, array_size * sizeof(int), hipMemcpyDeviceToDevice); 
        
    // compute array_index by prefix scan 
    prefix_scan(array_is_odd, array_index, array_size); 
        
    // get the number of odd numbers 
    hipMemcpy(num_odd, &array_index[array_size - 1], sizeof(int), hipMemcpyDeviceToHost); 
        
    // allocate GPU memory for the result array 
    int * array_device_out; 
    hipMalloc((void **) &array_device_out, (*num_odd) * sizeof(int)); 
        
    // collect the final result in GPU 
    get_odd<<<blocks, threads>>>(array_device, array_device_out, /* array_is_odd, */ array_index, array_size/* , *num_odd */); 
    hipDeviceSynchronize(); 
      
    // allocate CPU memory for the result array 
    int * array_o = (int *)malloc((*num_odd) * sizeof(int)); 
    
    // copy the result from GPU to CPU
    hipMemcpy(array_o, array_device_out, (*num_odd) * sizeof(int), hipMemcpyDeviceToHost); 
    
    // finish     
    hipFree(array_device_out); 
    hipFree(array_index); 
    hipFree(array_is_odd); 
    hipFree(array_device); 
    return array_o; 
}

/* 
* CPU main routine 
*/ 
int main(void) {
    // check device 
    check_dev(); 
    
    // data array on host 
    int array_size = 0; 
    int * array_i = read_data(&array_size); 
    
    // do compact & elasped time record
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int num_odd = 0; 
    hipEventRecord(start, 0);
    int * array_o = compact(array_i, &num_odd, array_size); 
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    // print to file 
    print_file(array_o, num_odd, "./q3.txt"); 
    
    // print debug information to stdout 
    printf(">> Number of odd numbers found: %d\n", num_odd); 
    printf(">> Average time elapsed: %f\n", elapsedTime);
    
    // finish 
    free(array_i); 
    free(array_o); 
    return 0; 
}
