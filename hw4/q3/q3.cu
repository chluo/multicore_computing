#include <stdio.h>
#include <stdlib.h>
#include <math.h> 
#include <hip/hip_runtime.h>

#define MAX_ARRAY_SIZE 1000000

/* -------------------------------------------------------------------------
  Algorithm description: 
  
  array_in      {1, 5, 3, 2, 6, 7, 9, 5, 3, 6} 
     |
	 |  parallel check odd/even: O(1) 
	\_/
  array_is_odd  {1, 1, 1, 0, 0, 1, 1, 1, 1, 0} 
     |
	 |  inclusive prefix scan: O(logN) 
	\_/
  array_index   {1, 2, 3, 3, 3, 4, 5, 6, 7, 7}
  
  num_odd := array_index[N - 1];                       -\
  array_o[num_odd];                                      |
  for i = 1 : N - 1 in parallel                          | O(1)
	if array_is_odd[i]                                   | 
		array_o[array_index[i] - 1] = array_in[i];       |
		                                               -/
  array_o       {1, 5, 3, 7, 9, 5, 3}
  
  T(N) = O(logN) 
------------------------------------------------------------------------- */

/* 
* Check GPU device 
*/ 
void check_dev(void) {
	int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        printf("!! Error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }
    int dev = 0;
    hipSetDevice(dev);
}

/*
* Calculate the number of threads per block based on array size 
*/ 
int calc_num_thread(int size) {
	int approx = (int)sqrt((double)size); 
	// find the nearest power of 2 
	int pow2 = 1; 
	while (pow2 < approx) {
		pow2 <<= 1; 
	}
	return pow2; 
}

/* 
* Read data from ./inp.txt 
* Return the pointer to the data array 
* Ouput the number of data items thru passed-in pointer (int * size)
*/ 
int * read_data(int * size) 
{
    FILE * fptr = fopen("./inp.txt", "r"); 
    if (!fptr) {
        printf("!! Error in opening data file \n"); 
        exit(1); 
    }
    int cur_array_size = MAX_ARRAY_SIZE; 
    int * buffer = (int *)malloc(cur_array_size * sizeof(int)); 
    
    int i = 0; 
    while (!feof(fptr)) {
        if (fscanf(fptr, "%d,", &buffer[i]) != 1) {
            printf("!! Error in importing data from file \n"); 
            exit(1); 
        }
        ++i;         
    }
    
	fclose(fptr); 
    *size = i; 
    return buffer; 
}

/* 
* Outputs the result array into file 
*/ 
void print_file(int * array, int array_size) {
	FILE * fptr_b = fopen("./q3.txt", "w"); 
    if (!fptr_b) {
        printf("!! Error in opening output file \n"); 
        exit(1);
    }
    for (int i = 0; i < array_size; ++i) {
        fprintf(fptr_b, "%d", array[i]); 
        if (i < array_size - 1) 
            fprintf(fptr_b, ", "); 
    }
    fclose(fptr_b); 
}

/* 
* GPU kernel: parallel odd/even check
* The output array has 1/odd or 0/even at the corresponding spot
*/ 
__global__ void odd_check(int * array_i, int * array_o, int array_size) {
	int myId = threadIdx.x + blockDim.x * blockIdx.x;
	if (myId < array_size) {
		array_o[myId] = array_i[myId] % 2;  
	}
}

/* 
* GPU kernel: inclusive prefix scan 
*/ 
__global__ void prefix_scan(int * array_i, int * array_o, int array_size) {
	// shared memory for intermediate results
	extern __shared__ int sdata[]; 
	
	int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int thId = threadIdx.x;
	
	// do scan in shared memory 
	int dist = 1; 
	while (dist < array_size) {
		if (!(myId < dist) && myId < array_size) {
			sdata[thId] += array_i[myId - dist]; 
		}
		__syncthreads();  
		dist *= 2; 
	}
	
	// copy the result to the output array 
	if (myId < array_size) {
		array_o[myId] = sdata[thId]; 
	}
}

/* 
* GPU kernel: compact the input array to get the odd numbers 
*/ 
__global__ void get_odd(int * array_i, int * array_o, int * array_is_odd, int * array_index, int array_size, int num_odd) {
	int myId = threadIdx.x + blockDim.x * blockIdx.x;
	if (myId < array_size) {
		if (array_is_odd[myId]) {
			array_o[array_index[myId] - 1] = array_i[myId]; 
		}
	}
}

/* 
* Compact algorithm: put the odd numbers in the input array into the output array
* Returns the pointer to the output array
* Ouputs the number of odd numbers thru passed-in pointer (int * num_odd) 
*/ 
int * compact(int * array_i, int * num_odd, int array_size) {
	// dynamically calculate the number of threads and blocks 
	const int maxThreadsPerBlock = calc_num_thread(array_size);
    int threads = maxThreadsPerBlock;
    int blocks = (array_size + maxThreadsPerBlock - 1) / maxThreadsPerBlock;
	
	// copy the input array into GPU shared memory 
	int * array_device; 
	hipMalloc((void **) &array_device, array_size * sizeof(int)); 
	hipMemcpy(array_device, array_i, array_size * sizeof(int), hipMemcpyHostToDevice); 
	
	// allocate GPU memories for array_is_odd and array_index 
	int * array_is_odd, * array_index; 
	hipMalloc((void **) &array_is_odd, array_size * sizeof(int)); 
	hipMalloc((void **) &array_index, array_size * sizeof(int)); 
	
	// compute array_is_odd 
	odd_check<<<blocks, threads>>>(array_device, array_is_odd, array_size); 
	
	// compute array_index by prefix scan 
	// prefix_scan<<<blocks, threads, threads * sizeof(int)>>>(array_is_odd, array_index, array_size); 
	
	// get the number of odd numbers 
	*num_odd = array_index[array_size - 1]; 
	
	// allocate GPU memory for the result array 
	int * array_device_out; 
	hipMalloc((void **) &array_device_out, (*num_odd) * sizeof(int)); 
	
	// compute the result
	get_odd<<<blocks, threads>>>(array_device, array_device_out, array_is_odd, array_index, array_size, *num_odd); 
	
	// allocate CPU memory for the result array 
	int * array_o = (int *)malloc((*num_odd) * sizeof(int)); 
	
	// copy the result from GPU to CPU
	hipMemcpy(array_o, array_device_out, (*num_odd) * sizeof(int), hipMemcpyDeviceToHost); 
	
	// finish 
	hipFree(array_device); 
	hipFree(array_device_out); 
	hipFree(array_is_odd); 
	hipFree(array_index); 
	return array_o; 
}

/* 
* CPU main routine 
*/ 
int main(void) {
	// check device 
	check_dev(); 
    
    // data array on host 
    int array_size = 0; 
    int * array_i = read_data(&array_size); 
	
	// do compact 
	int num_odd = 0; 
	int * array_o = compact(array_i, &num_odd, array_size); 
	
	// print to file 
	print_file(array_o, array_size); 
	
	// print debug information to stdout 
	printf(">> Number of odd numbers found: %d\n", num_odd); 
	
	// finish 
	free(array_i); 
	free(array_o); 
	return 0; 
}
